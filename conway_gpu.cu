#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include <stdlib.h>
#include <unistd.h>
#include "conway_gpu.h"
#include "util.h"

namespace cgl_gpu
{
    // In this case the rule set is declared outsid of class to be 
    // called in the kernel
    __host__ __device__
    int rules(int state, int n)
    {
        // Game rules        
        // Any live cell with two or three live neighbours survives.
        // Any dead cell with three live neighbours becomes a live cell.
        // All other live cells die in the next generation. Similarly, all other dead cells stay dead.
        int new_state = 0;
        
        if(state==1)
        {
            if(n==2 || n==3) new_state = 1;
            else new_state = 0;
        }
        else if(state==0)
        {
            if(n==3) new_state = 1;
            else new_state = 0;
        }
        return new_state;
    }

    namespace kernels 
    {
        __global__
        void update_state(int* x, std::size_t nx, std::size_t ny)
        {
            // Shared memory
            // extern __shared__ int buffer[];

            auto i = threadIdx.x + blockDim.x*blockIdx.x;
            auto j = threadIdx.y + blockDim.y*blockIdx.y;

            // Position in the array based on matrix coordinates
            size_t pos;
            // Number of neighbors alive
            int n_alive = 0;

            if(i>0 && i<nx-1 && j>0 && j<ny-1)
            {
                pos     = get_pos(i,j,ny);
                n_alive = x[pos-1]+x[pos+1]
                          +x[get_pos(i-1,j-1,ny)]+x[get_pos(i-1,j,ny)]+x[get_pos(i-1,j+1,ny)]
                          +x[get_pos(i+1,j-1,ny)]+x[get_pos(i+1,j,ny)]+x[get_pos(i+1,j+1,ny)];
                
                x[pos] = rules(x[pos],n_alive);
            }
            // __syncthreads();
        }
    }

    conway_gpu::conway_gpu()
    {
        // Define default parameters
        this->pad = 1;
        this->nx  = 20+2*this->pad;
        this->ny  = 20+2*this->pad;
        this->p_init   = 0.1;
        this->ind = '*';
        // Allocate grid on host
        this->grid_host   = this->allocate_grid("host");
        this->grid_device = this->allocate_grid("device");
        // Initialize grid on host
        this->initialize_grid(this->p_init);
    }

    conway_gpu::conway_gpu(std::size_t nx, std::size_t ny, int pad, float p_init)
    {
        // Define default parameters
        this->pad = pad;
        this->nx  = nx+2*pad;
        this->ny  = ny+2*pad;
        this->p_init = p_init;
        this->ind = '*';
        // Allocate grid
        this->grid_host   = this->allocate_grid("host");
        this->grid_device = this->allocate_grid("device");
        // Initialize grid
        this->initialize_grid(this->p_init);
    }

    conway_gpu::conway_gpu(std::size_t nx, std::size_t ny, int pad, float p_init, char ind)
    {
        // Define default parameters
        this->pad = pad;
        this->nx  = nx+2*pad;
        this->ny  = ny+2*pad;
        this->p_init = p_init;
        this->ind = ind;
        // Allocate grid on host
        this->grid_host   = this->allocate_grid("host");
        this->grid_device = this->allocate_grid("device");
        // Initialize grid
        this->initialize_grid(this->p_init);
    }

    conway_gpu::conway_gpu(std::size_t nx, std::size_t ny, int pad, int** init, char ind)
    {
        // Define default parameters
        this->pad = pad;
        this->nx  = nx+2*pad;
        this->ny  = ny+2*pad;
        this->ind = ind;
        // Allocate grid on host
        this->grid_host   = this->allocate_grid("host");
        this->grid_device = this->allocate_grid("device");
        // Initialize grid
        this->initialize_grid(init);
    }
    
    //____________________________________VISUALIZATION METHODS___________________________________//
    void conway_gpu::print_grid()
    {
        size_t pos;
        for(auto r=0; r<this->nx; r++) {
            for(auto c=0; c<this->ny; c++) {
                pos = get_pos(r,c,this->ny);
                if(this->grid_host[pos]) {
                    std::cout << this->ind;
                }
                else if(r==0 || r==this->nx-1 || c==0 || c==this->ny-1)
                {
                    std::cout << "x";
                }
                else 
                {
                    std::cout << " ";
                }
            }
            std::cout << "\n";
         }
    }
    
    //_________________________________________RUN METHODS________________________________________//
    void conway_gpu::run(int n_gens)
    {
        int gen = 0;
        while(gen<n_gens)
        {
            // Copy state to device
            this->update_device();
            // Update state on device
            this->update_state(this->grid_device,this->nx,this->ny);
            // Update grid on host
            this->update_host();
            gen++;
        }
    }

    // void conway_gpu::simulate(int n_gens)
    // {
    //     system("clear");
    //     int gen = 0;
    //     this->print_grid();
    //     while(gen<n_gens)
    //     {
    //         this->run(1);
    //         this->print_grid();
    //         sleep(1);
    //         system("clear");
    //         gen++;
    //     }
        
    // }

    int* conway_gpu::allocate_grid(const char* target)
    {
        int* ptr;
        if(target=="host") 
        {
            ptr = malloc_host<int>(this->nx*this->ny,0);
        }
        else if(target=="device") 
        {
            ptr = malloc_device<int>(this->nx*this->ny);
        }
        return ptr;
    }

    void conway_gpu::update_device()
    {
        copy_to_device<int>(this->grid_host,this->grid_device,this->nx*this->ny); 
    }

    void conway_gpu::update_host()
    {
        copy_to_host<int>(this->grid_device,this->grid_host,this->nx*this->ny); 
    }

    void conway_gpu::initialize_grid(float p_init)
    {
        // Random seed based on time 
        srand (time(NULL));

        size_t pos;
        for(auto r=this->pad; r<this->nx-this->pad; r++) {
            for(auto c=this->pad; c<this->ny-this->pad; c++) {
               pos = get_pos(r,c,this->ny);
               // std::cout<<pos<<"\n";
               if((float) rand()/RAND_MAX < p_init) this->grid_host[pos] = 1;
            }
        }
    }

    void conway_gpu::initialize_grid(int** init)
    {
        int pos = 0;
        for(auto r=0; r<this->nx-2*this->pad; r++) {
            for(auto c=0; c<this->ny-2*this->pad; c++) {
               pos = get_pos(r,c,this->ny);
               // this->grid[r+this->pad][c+this->pad] = init[r][c];
            }
        }
    }

    void conway_gpu::update_state(int* x, std::size_t nx, std::size_t ny)
    {
        // Testing
        dim3 block_dim(16,16);
        dim3 grid_dim((this->nx+block_dim.x-1)/block_dim.x,(this->ny+block_dim.y-1)/block_dim.y);
        kernels::update_state<<<grid_dim,block_dim>>>(this->grid_device,this->nx,this->ny);
    }
    // void conway_gpu::update_state()
    // {
    //     // Temporary grid
    //     int n_live   = 0;
    //     int** buffer = this->allocate_grid();

    //     for(int r=1; r<this->nx-1; r++) {
    //         for(int c=1; c<this->ny-1; c++) {
    //             n_live = this->check_neighbors(r,c);
    //             buffer[r][c] = this->rules(this->grid[r][c], n_live);
    //         }
    //     }

    //     // Copy buffer to grid
    //     for(int r=0; r<this->nx; r++) {
    //         for(int c=0; c<this->ny; c++) {
    //             this->grid[r][c] = buffer[r][c];
    //         }
    //     }

    //     free(buffer);
    // }

    // int conway_gpu::check_neighbors(int i, int j)
    // {
    //     int n = 0;
    //     for(auto r=i-1;r<=i+1;r++) 
    //     {
    //         for(auto c=j-1;c<=j+1;c++)
    //         {
    //             if(r==i && c==j) {
    //                 continue;
    //             }
    //             else {
    //                 n+=this->grid[r][c];
    //             }
    //         }
    //     }
    //     return n;
    // }

    // __host__ __device__
    // int conway_gpu::rules(int state, int n)
    // {
    //     // Game rules        
    //     // Any live cell with two or three live neighbours survives.
    //     // Any dead cell with three live neighbours becomes a live cell.
    //     // All other live cells die in the next generation. Similarly, all other dead cells stay dead.
    //     int new_state = 0;
        
    //     if(state==1)
    //     {
    //         if(n==2 || n==3) new_state = 1;
    //         else new_state = 0;
    //     }
    //     else if(state==0)
    //     {
    //         if(n==3) new_state = 1;
    //         else new_state = 0;
    //     }
    //     return new_state;
    // }

    //_________________________________________GET METHODS________________________________________//
    // int** conway_gpu::get_grid()
    // {
    //     return this->grid;
    // }

    // int conway_gpu::get_gridsize()
    // {
    //     return (this->nx-1)*(this->ny-1);
    // }

    // int conway_gpu::get_nx()
    // {
    //     return this->nx;
    // }

    // int conway_gpu::get_ny()
    // {
    //     return this->ny;
    // }

    // int conway_gpu::get_pad()
    // {
    //     return this->pad;
    // }

    // void conway_gpu::save(const char* filename)
    // {
    //     // Save results to file
    //     std::ofstream myfile;
    //     myfile.open(filename);

    //     for(auto r=0; r<this->nx; r++) {
    //         for(auto c=0; c<this->ny; c++) {
    //             if(c<this->ny-1) myfile << this->grid[r][c] << " ";
    //             else myfile << this->grid[r][c];
    //         }
    //         myfile << "\n";
    //     }
    //     myfile.close();
    // }
}
